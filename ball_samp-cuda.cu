
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <assert.h>
#include <string>
#include <iomanip>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>




__global__ void randgen(double* arr, int rows, int cols){
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	hiprandState state;
	hiprand_init(15, x, 0, &state);
	double randnum = hiprand_uniform(&state);
	double sum = 0;
	
	while(1){
	if (x<rows){
		sum = 0;

		for(int r=0; r<cols; r++){
			randnum = hiprand_uniform(&state);
			sum += randnum*randnum;

		}
		if(sum<1){
			arr[x] = sum;
			break;
		}
//		printf("%f\n", arr[index]);
	}
		}
}


__global__ void get_hists(double* res, double* hists, double*arr, int n, int size, int total_vol){
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	if(x<n){
		double sum = 0;
		for(int i=0; i<size; i++){
			if(res[i]>=arr[x] && res[i]<arr[x+1])
				sum += 1;
		}
		hists[x] = (double)sum/total_vol;	
	}
}


int main(int argc, char* argv[]){

	// **Note** The following variables in the editables can be changed. Select what works best with your system.
	// Editables {	
	int min_dim = 1; // Min dimension - start dimension
	int n = 1500000; // Number of points to sample for dim = min_dim
	float d = 1.5; // Number of points to sample decreases by a factor of d for each increament in dim 
	int max_dim = 16; // Max dimension - end dimension
	// }


	for(int dim=min_dim; dim<max_dim+1; dim++){

	int rows = n;	
	int cols = dim;
	
	size_t total_bytes = rows*sizeof(double);
	
	// Allocate memory for the matrix on host
	// double* mat = (double*)malloc(total_bytes); // host
	double* matd; // device


	// Allocate memory for the matrix on device
	hipMalloc(&matd, total_bytes);
	

	// copy host matrix to device (redundant! No need to because results are what we will obtain from the device. Also it is not initiated on host)

	// cudaMemcpy(matd, mat, total_bytes, cudaMemcpyHostToDevice);

	int blockSize, gridSize;

	// Number of threads in each block
	blockSize = 256;
	
	// Number of thread blocks in grid
	gridSize = (int)ceil((float)(rows/blockSize));
	// Execute the kernel
	randgen<<<gridSize, blockSize>>>(matd, rows, cols);

	// std::cout<<"done"<<std::endl;

	// Copy matrix back to host;
	// cudaMemcpy(mat, matd, total_bytes, cudaMemcpyDeviceToHost);

	// to print the initiated array	
	// for(int i=0; i<100; i++){
	//	std::cout<<mat[i]<<" ";	
	//}

	
	int size = rows; // size of results	
	// std::cout<<results[size-2]<<std::endl;
	
	int total_vol = n; // total number of points present inside the hypersphere
	
	
	// creating intervals
	int samp = 101;
	double* arr = (double*)malloc(samp*sizeof(double));
	arr[0] = 0;
	
	for(int i=1; i<samp; i++){
		arr[i] = arr[i-1] + 0.01;
	}

	double* arrd;
	hipMalloc(&arrd, samp*sizeof(double));
	hipMemcpy(arrd, arr, samp*sizeof(double), hipMemcpyHostToDevice);

	// obtaining hists
	double* hists = (double*)malloc((samp-1)*sizeof(double));
	double* histsd;
	hipMalloc(&histsd, (samp-1)*sizeof(double));
	
	blockSize = 256;
	gridSize = (int)ceil((float)(samp-1)/blockSize);
	
	get_hists<<<gridSize, blockSize>>>(matd, histsd, arrd, samp-1, size, total_vol);

	hipMemcpy(hists, histsd, (samp-1)*sizeof(double), hipMemcpyDeviceToHost);


	double total_sum = 0;
	std::cout<<"For dimension - "<<dim<<": "<<std::endl;
	std::cout<<"Number of points sampled = "<< total_vol<<std::endl;
	std::cout<<"Fractional volumes = { " ;
	for(int i=0; i<samp-1; i++){
		total_sum += hists[i];
		std::cout<<hists[i]<<", ";
	}
	std::cout<<" }";
	std::cout<<std::endl;
	std::cout<<"Total sum of fractions = "<<total_sum<<std::endl;
	std::cout<<std::endl;



	









	n /= d;

	hipDeviceSynchronize();








	// Release host memory
//	free(mat);
//	free(results);
	free(arr);
	free(hists);


	// Release device memory
	hipFree(matd);
	hipFree(histsd);
	hipFree(arrd);
	}
	return 0;
}

