
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <assert.h>
#include <string>
#include <iomanip>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>




__global__ void randgen(double* arr, int rows, int cols){
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	hiprandState state;
	hiprand_init(15, x, 0, &state);
	double randnum = hiprand_uniform(&state);
	double sum = 0;
	
	while(1){
	if (x<rows){
		sum = 0;

		for(int r=0; r<cols; r++){
			randnum = hiprand_uniform(&state);
			sum += randnum*randnum;

		}
		if(sum<1){
			arr[x] = sum;
			break;
		}
//		printf("%f\n", arr[index]);
	}
		}
}


__global__ void get_hists(double* res, double* hists, double*arr, int n, int size, int total_vol){
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	if(x<n){
		double sum = 0;
		for(int i=0; i<size; i++){
			if(res[i]>=arr[x] && res[i]<arr[x+1])
				sum += 1;
		}
		hists[x] = (double)sum/total_vol;	
	}
}


int main(int argc, char* argv[]){
	
//	int dim = std::stoi(argv[1]);
//	int dim = 3;
	int n = 15000000;
	for(int dim=2; dim<17; dim++){

	int rows = n;	
	int cols = dim;
	
	size_t total_bytes = rows*sizeof(double);
	
	// Allocate memory for the matrix on host
//	double* mat = (double*)malloc(total_bytes); // host
	double* matd; // device


	// Allocate memory for the matrix on device
	hipMalloc(&matd, total_bytes);
	

	// copy host matrix to device (redundant! No need to because results are what we will obtain from the device. Also it is not initiated on host)

//	cudaMemcpy(matd, mat, total_bytes, cudaMemcpyHostToDevice);

	int blockSize, gridSize;

	// Number of threads in each block
	blockSize = 256;
	
	// Number of thread blocks in grid
	gridSize = (int)ceil((float)(rows/blockSize));
	// Execute the kernel
	randgen<<<gridSize, blockSize>>>(matd, rows, cols);
//	std::cout<<"done"<<std::endl;

	// Copy matrix back to host;
//	cudaMemcpy(mat, matd, total_bytes, cudaMemcpyDeviceToHost);

	// to print the initiated array	
//	for(int i=0; i<100; i++){
//		std::cout<<mat[i]<<" ";
		
//	}

	
	int size = rows; // size of results	
//	std::cout<<results[size-2]<<std::endl;
	
	int total_vol = n; // total number of points present inside hypersphere
	
	
	// creating intervals
	int samp = 101;
	double* arr = (double*)malloc(samp*sizeof(double));
	arr[0] = 0;
	
	for(int i=1; i<samp; i++){
		arr[i] = arr[i-1] + 0.01;
	}

	double* arrd;
	hipMalloc(&arrd, samp*sizeof(double));
	hipMemcpy(arrd, arr, samp*sizeof(double), hipMemcpyHostToDevice);

	// obtaining hists
	double* hists = (double*)malloc((samp-1)*sizeof(double));
	double* histsd;
	hipMalloc(&histsd, (samp-1)*sizeof(double));
	
	blockSize = 256;
	gridSize = (int)ceil((float)(samp-1)/blockSize);
	
	get_hists<<<gridSize, blockSize>>>(matd, histsd, arrd, samp-1, size, total_vol);

	hipMemcpy(hists, histsd, (samp-1)*sizeof(double), hipMemcpyDeviceToHost);


//	for(int i=0; i<samp-1; i++){
//		int sum = 0;
//		for(int j=0; j<size; j++){
//			if(results[j]>=arr[i] && results[j]<arr[i+1])
//				sum += 1;
//		}
//		hists[i] = (double)sum/total_vol;
//	}

	double total_sum = 0;
	std::cout<<"For dimension - "<<dim<<": ";
	for(int i=0; i<samp-1; i++){
		total_sum += hists[i];
		std::cout<<hists[i]<<" ";
	}
	std::cout<<std::endl;
	std::cout<<std::endl;
	std::cout<<total_sum<<std::endl;



	









	n /= 2;

	hipDeviceSynchronize();








	// Release host memory
//	free(mat);
//	free(results);
	free(arr);
	free(hists);


	// Release device memory
	hipFree(matd);
	hipFree(histsd);
	hipFree(arrd);
	}
	return 0;
}

